#include "hip/hip_runtime.h"
﻿#define _USE_MATH_DEFINES
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""

using namespace std;

// Complex data type
typedef float2 Complex;

static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void TwiddleMult(Complex*, Complex*, int*);
static __global__ void TwiddleMult3d(Complex*, Complex*, int*);

//declaration

hipError_t fft_3d(int, int, int);
hipError_t test_3d_dec();
hipError_t fft_3d_rec(Complex*, int, int, int);



hipError_t cudaStatus = hipSetDevice(0);

int main()
{
    //test_3d_dec();
    fft_3d(64, 64, 65536);

    return 0;
}

hipError_t test_3d_dec() {
    int X = 2;
    int Y = 4;
    int Z = 8;
    printf("X = %d, Y = %d, Z = %d \n", X, Y, Z);
    int deg = (int)log2(Z);
    long long Z1 = (int)pow(2, deg / 2);
    long long Z2 = (int)pow(2, (deg + 1) / 2);

    long long  signalSize = X * Y * Z;

    // float2 2 * 4byte each element
    long long gpu_mem_size = X * Y * Z; // elemnts for 2GB
    long long gpu_mem_size_b = gpu_mem_size * sizeof(Complex); // bytes for 2GB 


    // Allocate host and device memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* d_signal;
    Complex* d_result;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size_b));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_result), gpu_mem_size_b));


    srand(3);
    // Initialize signal
    for (unsigned int i = 0; i < signalSize; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }


    //allocate memory for twiddle

    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * Z1 * Z2));
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), sizeof(Complex) * Z1 * Z2));

    //compute twidlle factors
    // exp(2 * pi * (m - 1) * (j - 1) / n), m - ñòðîêà, j - ñòîëáåö
    for (int i = 0; i < Z2; i++) {
        for (int j = 0; j < Z1; j++) {
            h_twiddle[i * Z1 + j].x = real(polar(1.0, -2 * M_PI * i * j / Z));
            h_twiddle[i * Z1 + j].y = imag(polar(1.0, -2 * M_PI * i * j / Z));
            // cout << "(" << h_twiddle[i * Z1 + j].x << "; " << h_twiddle[i * Z1 + j].y << ") ";
        }
        //  cout << "\n";
    }



    int C = Z1;
    int W = gpu_mem_size / C;

    int tC = C;
    int tW = W;

    int* h_vars = (int*)malloc(3 * sizeof(int));
    h_vars[0] = C; h_vars[1] = W, h_vars[2] = X * Y;

    int* d_vars;
    checkCudaErrors(hipMalloc(&d_vars, 3 * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_vars, h_vars, 3 * sizeof(int), hipMemcpyHostToDevice));


    checkCudaErrors(hipMemcpyAsync(d_signal, h_signal, gpu_mem_size_b, hipMemcpyHostToDevice));

    //transfer twiddle

    checkCudaErrors(hipMemcpy(d_twiddle, h_twiddle, sizeof(Complex) * Z1 * Z2,
        hipMemcpyHostToDevice));
    //make plan for d(Z1, XYZ2, XY)
    hipfftHandle plan_advZ1;

    int n[1] = { tC };
    int inembed[] = { C };
    int onembed[] = { W };
    int istride = tW;
    int idist = 1;
    int ostride = X * Y;
    int odist = 1;
    int batch = X * Y;
    // might need for loop (?) cos of possibility of output overlap
    // transpose by advanced layout
    checkCudaErrors(hipfftPlanMany(&plan_advZ1, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    for (int k = 0; k < tW / (X * Y); k++) {
        checkCudaErrors(hipfftExecC2C(plan_advZ1, reinterpret_cast<hipfftComplex*>(d_signal + (k * X * Y)),
            reinterpret_cast<hipfftComplex*>(d_result + (k * X * Y * tC)), HIPFFT_FORWARD));
    }
    hipDeviceSynchronize();


    dim3 threadsPerBlock(8, 8, 16);
    dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (W / (X * Y) + threadsPerBlock.y - 1) / threadsPerBlock.y,
        ((X * Y) + threadsPerBlock.z - 1) / threadsPerBlock.z);
    TwiddleMult3d << <numBlocks, threadsPerBlock >> > (d_result, d_twiddle, d_vars);
    hipDeviceSynchronize();

     //make plan for d(Y, X, X)

    hipfftHandle plan_advY;

    n[0] = Y;
    inembed[0] = tC;
    onembed[0] = tW;
    istride = X;
    idist = 1;
    ostride = X;
    odist = 1;
    batch = X;

    int h_C = C;
    checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    hipDeviceSynchronize();
    int tZ2 = tW / (X * Y);
    for (int k = 0; k < h_C * tZ2; k++) {
        checkCudaErrors(hipfftExecC2C(plan_advY, d_result + (k * X * Y),
            d_signal + (k * X * Y), HIPFFT_FORWARD));
    }
    hipDeviceSynchronize();

    C = Z2;
    W = gpu_mem_size / C;
    tC = C;
    tW = W;

    hipfftHandle plan_advZ2;

    n[0] = tC;
    inembed[0] = tC;
    onembed[0] = tW;
    istride = tW;
    idist = 1;
    ostride = tW;
    odist = 1;
    batch = tW;


    checkCudaErrors(hipfftPlanMany(&plan_advZ2, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    checkCudaErrors(hipfftExecC2C(plan_advZ2, reinterpret_cast<hipfftComplex*>(d_signal),
        reinterpret_cast<hipfftComplex*>(d_result), HIPFFT_FORWARD));


    hipfftHandle plan_advX;

    n[0] = X;
    inembed[0] = tC;
    onembed[0] = tW;
    istride = 1;
    idist = X;
    ostride = 1;
    odist = X;
    batch = tC * tW / X;

    cout << "Checking if output is right\n";
    checkCudaErrors(hipfftPlanMany(&plan_advX, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));

    checkCudaErrors(hipfftExecC2C(plan_advX, d_result,
        d_signal, HIPFFT_FORWARD));

    checkCudaErrors(hipMemcpy(h_result, d_signal, gpu_mem_size_b,
        hipMemcpyDeviceToHost));

    hipMemcpy(d_signal, h_signal, X * Y * Z * sizeof(Complex), hipMemcpyHostToDevice);
    hipfftHandle plan;
    hipfftPlan3d(&plan, Z, Y, X, HIPFFT_C2C);
    hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD);

    hipMemcpy(h_signal, d_signal, Z * Y * X * sizeof(Complex),
        hipMemcpyDeviceToHost);


    for (int i = 0; i < X * Y * Z; i++) {
        //cout << h_result[i].x << " ";
        if (abs(h_result[i].x - h_signal[i].x) > .01) {
            cout << h_result[i].x << " " << h_signal[i].x << " x " << i << "\n";
        }
        if (abs(h_result[i].y - h_signal[i].y) > .01) {
            cout << h_result[i].y << " " << h_signal[i].y << " y " << i << "\n";
        }
    }
    cout << "Done\n";
    return cudaStatus;
}
hipError_t fft_3d(int X,int Y, int Z) {

    printf("X = %d, Y = %d, Z = %d \n", X, Y, Z);

    int deg = (int)log2(Z);

    int Z1 = (int)pow(2, deg / 2);
    int Z2 = (int)pow(2, (deg + 1) / 2);
    int signalSize = X * Y * Z;

    // float2 2 * 4byte each element
    int gpu_mem_size = 1048576; // 
    long long gpu_mem_size_b = gpu_mem_size * sizeof(Complex); // bytes


    // Allocate host and device memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result_2 = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* buffer = reinterpret_cast<Complex*>(malloc(gpu_mem_size_b));

    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size_b));   
    Complex* d_result;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_result), gpu_mem_size_b));

    
    // Initialize signal
    srand(3);
    for (unsigned int i = 0; i < signalSize; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }

  
    //allocate memory for twiddle

    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * Z1 * Z2));
    
    //compute twidlle factors
    // exp(2 * pi * (m - 1) * (j - 1) / n), m - строка, j - столбец
    for (int i = 0; i < Z2; i++) {
        for (int j = 0; j < Z1; j++) {
            h_twiddle[i * Z1 + j].x = (float)real(polar(1.0, -2 * M_PI * i * j / Z));
            h_twiddle[i * Z1 + j].y = (float)imag(polar(1.0, -2 * M_PI * i * j / Z));
        }
    }
    //streams might need later
    const int sNum = 2;
    hipStream_t stream[sNum];
    for (int i = 0; i < sNum; ++i)
        hipStreamCreate(&stream[i]);


    cout <<"Z1 = "<< Z1 << ", Z2 = " << Z2 << "\n";

    int C = Z1;
    int W = gpu_mem_size / C;
    int tZ2 = W / (X * Y);

    
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), C * W / (X * Y) * sizeof(Complex)));

    // variables for kernel {W, C, X*Y}
    int* h_vars = (int*)malloc(3 * sizeof(int));
    h_vars[0] = C; h_vars[1] = W, h_vars[2] = X * Y;

    int* d_vars;
    checkCudaErrors(hipMalloc(&d_vars, 3 * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_vars, h_vars, 3 * sizeof(int), hipMemcpyHostToDevice));


    // X * Y fits into memory
    int it_s = X * Y * Z2 / W;
   // cout << it_s << "\n";
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    float trans = 0.0, twiddle = 0.0, fft = 0.0;

    //make plan for d(Z1, XYZ2, XY)
    hipfftHandle plan_advZ1;

    int n[1] = { C };
    int inembed[] = { C };
    int onembed[] = { W };
    int istride = W;
    int idist = 1;
    int ostride = X * Y;
    int odist = 1;
    int batch = X * Y;

    // transpose by advanced layout
    checkCudaErrors(hipfftPlanMany(&plan_advZ1, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));

    //make plan for d(Y, X, X)
    hipfftHandle plan_advY;

    n[0] = Y;
    inembed[0] = C;
    onembed[0] = W;
    istride = X;
    idist = 1;
    ostride = X;
    odist = 1;
    batch = X;

    checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));

    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        hipEvent_t start_trans, stop_trans;
        hipEventCreate(&start_trans);
        hipEventCreate(&stop_trans);
        hipEventRecord(start_trans);

        for (unsigned int j = 0; j < C; j++) {
            if (j == C / 2)
                checkCudaErrors(hipMemcpyAsync(d_signal, buffer, C / 2 * W * sizeof(Complex), hipMemcpyHostToDevice, stream[i % sNum]));
            memcpy(buffer + (j * W), h_signal + (i * W) + (j * X * Y * Z2), W * sizeof(Complex));
            
        }
        checkCudaErrors(hipMemcpyAsync(d_signal + C / 2 * W , buffer + C / 2 * W, C / 2 * W * sizeof(Complex), hipMemcpyHostToDevice, stream[i % sNum]));
        // checkCudaErrors(hipMemcpyAsync(d_signal, buffer, C * W * sizeof(Complex), hipMemcpyHostToDevice));
        

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //transfer twiddle
        // W / (X*Y) is Z2 / const
        
        checkCudaErrors(hipMemcpyAsync(d_twiddle, h_twiddle + (i * C * tZ2), C * tZ2 * sizeof(Complex),
            hipMemcpyHostToDevice, stream[i % sNum]));

        hipEventRecord(stop_trans);
        hipEventSynchronize(stop_trans);
        float ms = 0;
        hipEventElapsedTime(&ms, start_trans, stop_trans);

        trans += ms;

        hipEventDestroy(start_trans);
        hipEventDestroy(stop_trans);

        hipEvent_t start_fft, stop_fft;
        hipEventCreate(&start_fft);
        hipEventCreate(&stop_fft);
        hipEventRecord(start_fft);
        checkCudaErrors(hipfftSetStream(plan_advZ1, stream[i % sNum]));

        for (int k = 0; k < W / (X * Y); k++) {
            checkCudaErrors(hipfftExecC2C(plan_advZ1, reinterpret_cast<hipfftComplex*>(d_signal + (k * X * Y)),
                reinterpret_cast<hipfftComplex*>(d_result + (k * X * Y * C)), HIPFFT_FORWARD));
        }
        checkCudaErrors(hipDeviceSynchronize());
        
        hipEventRecord(stop_fft);
        hipEventSynchronize(stop_fft);
        ms = 0;
        hipEventElapsedTime(&ms, start_fft, stop_fft);

        fft += ms;

        hipEventDestroy(start_fft);
        hipEventDestroy(stop_fft);

        hipEvent_t start_twidlle, stop_twidlle;
        hipEventCreate(&start_twidlle);
        hipEventCreate(&stop_twidlle);
        hipEventRecord(start_twidlle);

        //multiply by twiddle factor
        dim3 threadsPerBlock(8, 8, 16);
        dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (tZ2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
            ((X * Y) + threadsPerBlock.z - 1) / threadsPerBlock.z);
        TwiddleMult3d << <numBlocks, threadsPerBlock, 0 , stream[i % sNum] >> > (d_result, d_twiddle, d_vars);
        checkCudaErrors(hipDeviceSynchronize());

        hipEventRecord(stop_twidlle);
        hipEventSynchronize(stop_twidlle);
        ms = 0;
        hipEventElapsedTime(&ms, start_twidlle, stop_twidlle);

        twiddle += ms;

        hipEventDestroy(start_twidlle);
        hipEventDestroy(stop_twidlle);

        hipEventCreate(&start_fft);
        hipEventCreate(&stop_fft);
        hipEventRecord(start_fft);
        checkCudaErrors(hipfftSetStream(plan_advY, stream[i % sNum]));
        for (int k = 0; k < C * tZ2; k++) {
            checkCudaErrors(hipfftExecC2C(plan_advY, d_result + (k * X * Y),
                d_signal + (k * X * Y), HIPFFT_FORWARD));
        }
        checkCudaErrors(hipDeviceSynchronize());
        
        hipEventRecord(stop_fft);
        hipEventSynchronize(stop_fft);
        ms = 0;
        hipEventElapsedTime(&ms, start_fft, stop_fft);

        fft += ms;

        hipEventDestroy(start_fft);
        hipEventDestroy(stop_fft);

        
        hipEventCreate(&start_trans);
        hipEventCreate(&stop_trans);
        hipEventRecord(start_trans);

        //transport to host
        checkCudaErrors(hipMemcpyAsync(h_result + (i * C * W), d_signal, gpu_mem_size_b,
            hipMemcpyDeviceToHost, stream[i % sNum]));

        hipEventRecord(stop_trans);
        hipEventSynchronize(stop_trans);
        
        hipEventElapsedTime(&ms, start_trans, stop_trans);

        trans += ms;

        hipEventDestroy(start_trans);
        hipEventDestroy(stop_trans);
       

    }

    hipfftDestroy(plan_advZ1);
    hipfftDestroy(plan_advY);

    C = Z2;
    W = gpu_mem_size / C;
    it_s = X * Y * Z1 / W;

    //make plan for d(Z2, XYZ1, XYZ1)
    hipfftHandle plan_advZ2;

    n[0] = C;
    istride = W;
    idist = 1;
    ostride = W;
    odist = 1;
    batch = W;

    checkCudaErrors(hipfftPlanMany(&plan_advZ2, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));

    //make plan for d(X, 1, 1)
    hipfftHandle plan_advX;

    n[0] = X;
    istride = 1;
    idist = X;
    ostride = 1;
    odist = X;
    batch = C * W / X;


    checkCudaErrors(hipfftPlanMany(&plan_advX, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));


    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        hipEvent_t start_trans, stop_trans;
        hipEventCreate(&start_trans);
        hipEventCreate(&stop_trans);
        hipEventRecord(start_trans);

        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_result + (i * W) + (j * X * Y * Z1), W * sizeof(Complex));
        }
        checkCudaErrors(hipMemcpyAsync(d_signal, buffer, C * W * sizeof(Complex), hipMemcpyHostToDevice, stream[i % sNum]));

        hipEventRecord(stop_trans);
        hipEventSynchronize(stop_trans);
        float ms = 0;
        hipEventElapsedTime(&ms, start_trans, stop_trans);

        trans += ms;

        hipEventDestroy(start_trans);
        hipEventDestroy(stop_trans);

        hipEvent_t start_fft, stop_fft;
        hipEventCreate(&start_fft);
        hipEventCreate(&stop_fft);
        hipEventRecord(start_fft);
        checkCudaErrors(hipfftSetStream(plan_advZ2, stream[i % sNum]));

        checkCudaErrors(hipfftExecC2C(plan_advZ2, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_result), HIPFFT_FORWARD));
        hipEventRecord(stop_fft);
        hipEventSynchronize(stop_fft);
        ms = 0;
        hipEventElapsedTime(&ms, start_fft, stop_fft);

        fft += ms;

        hipEventDestroy(start_fft);
        hipEventDestroy(stop_fft);
        
        hipEventCreate(&start_fft);
        hipEventCreate(&stop_fft);
        hipEventRecord(start_fft);
        checkCudaErrors(hipfftSetStream(plan_advX, stream[i % sNum]));
        checkCudaErrors(hipfftExecC2C(plan_advX, d_result,
             d_signal, HIPFFT_FORWARD));

        hipEventRecord(stop_fft);
        hipEventSynchronize(stop_fft);
        ms = 0;
        hipEventElapsedTime(&ms, start_fft, stop_fft);

        fft += ms;

        hipEventDestroy(start_fft);
        hipEventDestroy(stop_fft);


        hipEventCreate(&start_trans);
        hipEventCreate(&stop_trans);
        hipEventRecord(start_trans);

        checkCudaErrors(hipMemcpyAsync(buffer, d_signal, C * W * sizeof(Complex),
            hipMemcpyDeviceToHost, stream[i % sNum]));
        
        for (unsigned int j = 0; j < C; j++) {
            memcpy(h_result_2 + (i * W) + (j * X * Y * Z1), buffer + (j * W), W * sizeof(Complex));
        }
        
        hipEventRecord(stop_trans);
        hipEventSynchronize(stop_trans);
        ms = 0;
        hipEventElapsedTime(&ms, start_trans, stop_trans);
        trans += ms;

        hipEventDestroy(start_trans);
        hipEventDestroy(stop_trans);
       
    }

    hipfftDestroy(plan_advZ2);
    hipfftDestroy(plan_advX);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "3d dec time, ms = " << milliseconds<<"\n";
    cout << "transport time  = " << trans<<"\n";
    cout << "twiddle time  = " << twiddle<<"\n";
    cout << "fft's time  = " << fft<<"\n";
    /*
   // for (int i = 0; i < signalSize; i++) {
    //    cout << h_result_2[i].x << " " << h_result_2[i].y << "\n";
    //}
    
    checkCudaErrors(hipFree(d_signal));
    
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), X* Y* Z * sizeof(Complex)));
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    checkCudaErrors(hipMemcpy(d_signal, h_signal, X* Y* Z * sizeof(Complex), hipMemcpyHostToDevice));
    hipfftHandle plan;



    checkCudaErrors(hipfftPlan3d(&plan, Z, Y, X, HIPFFT_C2C));
    checkCudaErrors(hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD));
    checkCudaErrors( hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_signal, d_signal, Z* Y* X * sizeof(Complex),
        hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "cufft time, ms = " << milliseconds << "\n";

    double rs = 0, is = 0;
    double rs2 = 0, is2 = 0;

    for (int i = 0; i < X * Y * Z; i++) {
        //cout << h_result[i].x << " ";
        if (abs(h_result_2[i].x - h_signal[i].x) > 1 ) {
            cout << h_result_2[i].x << " " << h_signal[i].x << " x " << i << "\n";
        }
        if (abs(h_result_2[i].y - h_signal[i].y) > 1 ) {
            cout << h_result_2[i].y << " " << h_signal[i].y << " y " << i << "\n";
        }
    }
    
    */
    free(h_signal);
    free(h_result);
    free(h_twiddle);
    hipFree(d_result);
    hipFree(d_signal);
    hipFree(d_twiddle);

    return cudaStatus;
}

hipError_t fft_3d_rec(Complex* h_signal, int X, int Y, int Z) {

    int deg = (int)log2(Z);

    int Z1 = (int)pow(2, deg / 2);
    int Z2 = (int)pow(2, (deg + 1) / 2);
    int signalSize = X * Y * Z;

    // float2 2 * 4byte each element
    int gpu_mem_size = 1048576; // 
    long long gpu_mem_size_b = gpu_mem_size * sizeof(Complex); // bytes


    // Allocate host and device memory for the signal
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result_2 = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size_b));
    Complex* d_result;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_result), gpu_mem_size_b));


    //allocate memory for twiddle
    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * Z1 * Z2));

    //compute twidlle factors
    // exp(2 * pi * (m - 1) * (j - 1) / n), m - строка, j - столбец
    for (int i = 0; i < Z2; i++) {
        for (int j = 0; j < Z1; j++) {
            h_twiddle[i * Z1 + j].x = (float)real(polar(1.0, -2 * M_PI * i * j / Z));
            h_twiddle[i * Z1 + j].y = (float)imag(polar(1.0, -2 * M_PI * i * j / Z));
        }
    }


    cout << "Z1 = " << Z1 << ", Z2 = " << Z2 << "\n";

    int C = Z1;
    int W = gpu_mem_size / C;


    Complex* buffer = reinterpret_cast<Complex*>(malloc(gpu_mem_size_b));
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), C * W / (X * Y) * sizeof(Complex)));
    // W, C, X*Y
    int* h_vars = (int*)malloc(3 * sizeof(int));
    h_vars[0] = C; h_vars[1] = W, h_vars[2] = X * Y;

    int* d_vars;
    checkCudaErrors(hipMalloc(&d_vars, 3 * sizeof(int)));
    checkCudaErrors(hipMemcpy(d_vars, h_vars, 3 * sizeof(int), hipMemcpyHostToDevice));


    // X * Y fits into memory
    int it_s = X * Y * Z2 / W;
    // cout << it_s << "\n";
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    int tZ2 = W / (X * Y);

    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_signal + (i * W) + (j * X * Y * Z2), W * sizeof(Complex));
        }
        checkCudaErrors(hipMemcpy(d_signal, buffer, C * W * sizeof(Complex), hipMemcpyHostToDevice));


        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //transfer twiddle
        // W / (X*Y) is Z2 / const

        checkCudaErrors(hipMemcpy(d_twiddle, h_twiddle + (i * C * tZ2), C * tZ2 * sizeof(Complex),
            hipMemcpyHostToDevice));

        //make plan for d(Z1, XYZ2, XY)
        hipfftHandle plan_advZ1;

        int n[1] = { C };
        int inembed[] = { C };
        int onembed[] = { W };
        int istride = W;
        int idist = 1;
        int ostride = X * Y;
        int odist = 1;
        int batch = X * Y;

        // transpose by advanced layout
        checkCudaErrors(hipfftPlanMany(&plan_advZ1, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));

        for (int k = 0; k < W / (X * Y); k++) {
            checkCudaErrors(hipfftExecC2C(plan_advZ1, reinterpret_cast<hipfftComplex*>(d_signal + (k * X * Y)),
                reinterpret_cast<hipfftComplex*>(d_result + (k * X * Y * C)), HIPFFT_FORWARD));
        }
        checkCudaErrors(hipDeviceSynchronize());
        hipfftDestroy(plan_advZ1);

        //multiply by twiddle factor
        dim3 threadsPerBlock(8, 8, 16);
        dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (tZ2 + threadsPerBlock.y - 1) / threadsPerBlock.y,
            ((X * Y) + threadsPerBlock.z - 1) / threadsPerBlock.z);
        TwiddleMult3d << <numBlocks, threadsPerBlock >> > (d_result, d_twiddle, d_vars);
        checkCudaErrors(hipDeviceSynchronize());

        //make plan for d(Y, X, X)
        hipfftHandle plan_advY;

        n[0] = Y;
        inembed[0] = C;
        onembed[0] = W;
        istride = X;
        idist = 1;
        ostride = X;
        odist = 1;
        batch = X;

        checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));

        for (int k = 0; k < C * tZ2; k++) {
            checkCudaErrors(hipfftExecC2C(plan_advY, d_result + (k * X * Y),
                d_signal + (k * X * Y), HIPFFT_FORWARD));
        }
        checkCudaErrors(hipDeviceSynchronize());
        hipfftDestroy(plan_advY);

        //transport to host
        checkCudaErrors(hipMemcpy(h_result + (i * C * W), d_signal, gpu_mem_size_b,
            hipMemcpyDeviceToHost));

    }


    C = Z2;
    W = gpu_mem_size / C;
    it_s = X * Y * Z1 / W;

    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_result + (i * W) + (j * X * Y * Z1), W * sizeof(Complex));
        }
        checkCudaErrors(hipMemcpyAsync(d_signal, buffer, C * W * sizeof(Complex), hipMemcpyHostToDevice));

        //make plan for d(Z2, XYZ1, XYZ1)
        hipfftHandle plan_advZ2;

        int n[1] = { C };
        int inembed[] = { C };
        int onembed[] = { W };
        int istride = W;
        int idist = 1;
        int ostride = W;
        int odist = 1;
        int batch = W;

        checkCudaErrors(hipfftPlanMany(&plan_advZ2, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_advZ2, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_result), HIPFFT_FORWARD));
        hipfftDestroy(plan_advZ2);

        //make plan for d(X, 1, 1)
        hipfftHandle plan_advX;

        n[0] = X;
        inembed[0] = C;
        onembed[0] = W;
        istride = 1;
        idist = X;
        ostride = 1;
        odist = X;
        batch = C * W / X;


        checkCudaErrors(hipfftPlanMany(&plan_advX, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));

        checkCudaErrors(hipfftExecC2C(plan_advX, d_result,
            d_signal, HIPFFT_FORWARD));
        hipfftDestroy(plan_advX);
        checkCudaErrors(hipMemcpy(buffer, d_signal, C * W * sizeof(Complex),
            hipMemcpyDeviceToHost));

        for (unsigned int j = 0; j < C; j++) {
            memcpy(h_result_2 + (i * W) + (j * X * Y * Z1), buffer + (j * W), W * sizeof(Complex));
        }


    }

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    cout << "3d dec time, ms = " << milliseconds << "\n";
    /*
   // for (int i = 0; i < signalSize; i++) {
    //    cout << h_result_2[i].x << " " << h_result_2[i].y << "\n";
    //}

    checkCudaErrors(hipFree(d_signal));

    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), X* Y* Z * sizeof(Complex)));
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    checkCudaErrors(hipMemcpy(d_signal, h_signal, X* Y* Z * sizeof(Complex), hipMemcpyHostToDevice));
    hipfftHandle plan;



    checkCudaErrors(hipfftPlan3d(&plan, Z, Y, X, HIPFFT_C2C));
    checkCudaErrors(hipfftExecC2C(plan, d_signal, d_signal, HIPFFT_FORWARD));
    checkCudaErrors( hipDeviceSynchronize());

    checkCudaErrors(hipMemcpy(h_signal, d_signal, Z* Y* X * sizeof(Complex),
        hipMemcpyDeviceToHost));
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    cout << "cufft time, ms = " << milliseconds << "\n";

    double rs = 0, is = 0;
    double rs2 = 0, is2 = 0;

    for (int i = 0; i < X * Y * Z; i++) {
        //cout << h_result[i].x << " ";
        if (abs(h_result_2[i].x - h_signal[i].x) > 1 ) {
            cout << h_result_2[i].x << " " << h_signal[i].x << " x " << i << "\n";
        }
        if (abs(h_result_2[i].y - h_signal[i].y) > 1 ) {
            cout << h_result_2[i].y << " " << h_signal[i].y << " y " << i << "\n";
        }
    }
    */

    free(h_signal);
    hipFree(d_result);
    hipFree(d_signal);
    hipFree(d_twiddle);

    return cudaStatus;
}

////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex pointwise multiplication
static __global__ void TwiddleMult(Complex* X, Complex* twiddle, int* d_vars) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int W = d_vars[1];
    int C = d_vars[0];
    if (i < C && j < W)
        X[j*C + i] = ComplexMul(X[j * C + i], twiddle[i*W + j]);
}
static __global__ void TwiddleMult3d(Complex* X, Complex* twiddle, int* d_vars) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    int xy = d_vars[2];
    int W = d_vars[1];
    int C = d_vars[0];

   // int Z2 = (k + xy * (i + C * j)) / (xy * C);

    if (i < C && j < W / xy && k < xy) {
        X[k + xy * (i + C * j)] = ComplexMul(X[k + xy * (i + C * j)], twiddle[i + j * C]);
        //X[k + xy * (i + C * j)].x = xy;
        //X[k + xy * (i + C * j)].y = k;
    }
}