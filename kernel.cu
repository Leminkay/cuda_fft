#include "hip/hip_runtime.h"
#define _USE_MATH_DEFINES
// includes, system
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <complex>

// includes, project
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include ""

using namespace std;

// Complex data type
typedef float2 Complex;
//8byte float
static __device__ __host__ inline Complex ComplexAdd(Complex, Complex);
static __device__ __host__ inline Complex ComplexScale(Complex, float);
static __device__ __host__ inline Complex ComplexMul(Complex, Complex);
static __global__ void TwiddleMult(Complex*, Complex*);
static __global__ void TwiddleMult(Complex*, Complex*, int);
__managed__ int C;
__managed__ int W;
__device__ int XY;



//declaration
hipError_t fft_1d(long long);
hipError_t fft_2d(long long, long long);
hipError_t fft_3d();
hipError_t test();
hipError_t test_3d();

hipError_t cudaStatus = hipSetDevice(0);

int main()
{

    
  //  if (cudaStatus != hipSuccess) {
    //test();
    test_3d();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
   // cudaStatus = hipDeviceReset();


    return 0;
}

// 
hipError_t test()
{
    int signalSize = 32;
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));

    Complex* d_signal;
    //Complex* d_result;
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), sizeof(Complex) * signalSize));
   // checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_result), sizeof(Complex) * signalSize));
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), sizeof(Complex) * signalSize));


    srand(2);
    for (unsigned int i = 0; i < signalSize; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }
    cout << "signal\n";
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            cout <<"(" <<h_signal[i * 4 + j].x << ", "<<h_signal[i*4 + j].y<<") ";
        }
        cout << "\n";
    }
    cout << "\n";
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            cout << h_signal[i * 4 + j].x<<"\n";
        }
        
    }
    cout << "\n";
    int X1 = 8, X2 = 4;
    
    for (int i = 0; i < X1  ; i++) {
        for (int j = 0; j < X2; j++) {
            h_twiddle[i * X2 + j].x = real(polar(1.0, -2 * M_PI * i * j / signalSize));
            h_twiddle[i * X2 + j].y = imag(polar(1.0, -2 * M_PI * i * j / signalSize));
        }
    }
    cout << "\n";
    cout << "twiddle\n";
    
    for (int i = 0; i < 8; i++) {
        for (int j = 0; j < 4; j++) {
            cout << "(" << h_twiddle[i * 4 + j].x << ", " << h_twiddle[i * 4 + j].y << ") ";
        }
        cout << "\n";
    }
    checkCudaErrors(hipMemcpy(d_twiddle, h_twiddle, sizeof(Complex) * signalSize,
        hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_signal, h_signal, sizeof(Complex) * signalSize, hipMemcpyHostToDevice));
    C = 8;
    W = 4;

    hipfftHandle plan_adv2;
    

    int n[1] = { C };
    int inembed[] = { C };
    int onembed[] = { W };
    int istride = W;
    int idist = 1;
    int ostride = 1;
    int odist = C;
    int batch = W;

    checkCudaErrors(hipfftPlanMany(&plan_adv2, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    checkCudaErrors(hipfftExecC2C(plan_adv2, reinterpret_cast<hipfftComplex*>(d_signal),
        reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
    checkCudaErrors(hipMemcpy(h_result, d_signal, sizeof(Complex) * signalSize,
        hipMemcpyDeviceToHost));
    
    cout << "\n";
    cout << "first round fft\n";
    for (int i = 0; i < X2; i++) {
        for (int j = 0; j < X1; j++) {
            cout << "(" << h_result [i * X1 + j] .x << ", " << h_result[i * X1 + j].y << ") ";
        }
        cout << "\n";
    }

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (W + threadsPerBlock.y - 1) / threadsPerBlock.y);
    TwiddleMult <<<numBlocks, threadsPerBlock >>> (d_signal, d_twiddle);
    checkCudaErrors(hipMemcpy(h_result, d_signal, sizeof(Complex) * signalSize,
        hipMemcpyDeviceToHost));
   
    cout << "\n";
    cout << "mult by twiddle\n";
    for (int i = 0; i < X2; i++) {
        for (int j = 0; j < X1; j++) {
            cout << "(" << h_result[i * X1 + j].x << ", " << h_result[i * X1 + j].y << ") ";
        }
        cout << "\n";
    }

    
    hipfftHandle plan_adv;


    n[0] = { W };
    inembed[0] = { C };
    onembed[0] = { C };
    istride = C;
    idist = 1;
    ostride = C;
    odist = 1;
    batch = C;
    checkCudaErrors(hipfftPlanMany(&plan_adv, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    checkCudaErrors(hipfftExecC2C(plan_adv, reinterpret_cast<hipfftComplex*>(d_signal),
        reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
    checkCudaErrors(hipMemcpy(h_result, d_signal, sizeof(Complex) * signalSize,
        hipMemcpyDeviceToHost));
    
    cout << "\n";
    cout << "resultt\n";
    for (int i = 0; i < X1 * X2; i++) {
        
        cout << "(" << h_result[i].x<< ", " << h_result[i].y << ") \n";
        
    }
    hipFree(d_signal);
    hipFree(d_twiddle);
    free(h_signal);
    free(h_result);
    free(h_twiddle);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    

Error:

    return cudaStatus;
}
hipError_t test_3d() {
    int signalSizeX = 8;
    int signalSizeY = 2;
    int signalSizeZ = 4;

    // Allocate host memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSizeY * signalSizeX * signalSizeZ));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSizeY * signalSizeX * signalSizeZ));
    srand(3);
    // Initialize the memory for the signal
    for (unsigned int i = 0; i < signalSizeY * signalSizeX * signalSizeZ; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }

    for (int z = 0; z < signalSizeZ; z++) {
        for (int y = 0; y < signalSizeY; y++) {
            for (int x = 0; x < signalSizeX; x++) {

                cout << "(" << h_signal[x + signalSizeX * (y + signalSizeY * z)].x << "; " << h_signal[x + signalSizeX * (y + signalSizeY * z)].y << ") ";
            }
            cout << "\n";

        }
        cout << "\n\n";
    }

    
    long long gpu_mem_size = signalSizeY * signalSizeX * signalSizeZ; 
    long long gpu_mem_size_b = signalSizeY * signalSizeX * signalSizeZ * sizeof(Complex); 

    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size_b));


    //assume that X * Y can fit in gpu
    int tC = signalSizeZ;
    int tW = signalSizeY * signalSizeX;
    Complex* buffer = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * tC * tW));


   
   
        //transfer data to gpu
    
  //  memcpy(buffer, h_signal, C * W * sizeof(Complex));
    checkCudaErrors(hipMemcpy(d_signal, h_signal, tC * tW * sizeof(Complex), hipMemcpyHostToDevice));
    

    //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
    //                           hipMemcpyHostToDevice));

    //transfer twiddle


    //make plan for z fft
    hipfftHandle plan_advZ;

    int n[1] = { tC };
    int inembed[] = { tW };
    int onembed[] = { tW };
    int istride = tW;
    int idist = 1;
    int ostride = tW;
    int odist = 1;
    int batch = tW;

    //checkCudaErrors(hipfftCreate(&plan_adv));
    //it is transpose
    checkCudaErrors(hipfftPlanMany(&plan_advZ, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    checkCudaErrors(hipfftExecC2C(plan_advZ, reinterpret_cast<hipfftComplex*>(d_signal),
        reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_result, d_signal, signalSizeX * signalSizeZ * signalSizeY * sizeof(Complex),
        hipMemcpyDeviceToHost));


    cout << "z fft\n";
    for (int z = 0; z < signalSizeZ; z++) {
        for (int y = 0; y < signalSizeY; y++) {
            for (int x = 0; x < signalSizeX; x++) {

                cout << "(" << h_result[x + signalSizeX * (y + signalSizeY * z)].x << "; " << h_result[x + signalSizeX * (y + signalSizeY * z)].y << ") ";
            }
            cout << "\n";

        }
        cout << "\n\n\n";
    }

    hipfftHandle plan_advX;

    // X fft
    n[0] = (int)signalSizeX;
    inembed[0] = tW;
    onembed[0] = tW;
    istride = 1;
    idist = (int)signalSizeX;
    ostride = 1;
    odist = (int)signalSizeX;
    batch = signalSizeY * signalSizeZ;
    //transport to host

    checkCudaErrors(hipfftPlanMany(&plan_advX, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    checkCudaErrors(hipfftExecC2C(plan_advX, reinterpret_cast<hipfftComplex*>(d_signal),
        reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipMemcpy(h_result, d_signal, signalSizeX * signalSizeZ * signalSizeY * sizeof(Complex),
        hipMemcpyDeviceToHost));

    cout << "x fft\n";
    for (int z = 0; z < signalSizeZ; z++) {
        for (int y = 0; y < signalSizeY; y++) {
            for (int x = 0; x < signalSizeX; x++) {

                cout << "(" << h_result[x + signalSizeX * (y + signalSizeY * z)].x << "; " << h_result[x + signalSizeX * (y + signalSizeY * z)].y << ") ";
            }
            cout << "\n";

        }
        cout << "\n\n";
    }


    
    // fft size y
    tC = signalSizeY;
    tW = gpu_mem_size / tC;


    //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
    //                           hipMemcpyHostToDevice));

    //make plan
    hipfftHandle plan_advY;
    n[0] = signalSizeY;
    
    inembed[0] = tW ;
    onembed[0] = tW;
    istride = signalSizeX;
    idist = 1;
    ostride = signalSizeX;
    odist = 1;
    batch = signalSizeX;

    //checkCudaErrors(hipfftCreate(&plan_adv));
    //does it transpose? (stride, dist)
    checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
        onembed, ostride, odist, HIPFFT_C2C, batch));
    for (int k = 0; k < signalSizeZ; k++) {
        ///cout << d_signal + (k * signalSizeX * signalSizeZ) << "\n";
        checkCudaErrors(hipfftExecC2C(plan_advY, d_signal + (k * signalSizeX * signalSizeY),
            d_signal + (k * signalSizeX * signalSizeY), HIPFFT_FORWARD));
    }
    //checkCudaErrors(hipDeviceSynchronize());
    //transport to host

    checkCudaErrors(hipMemcpy(h_result, d_signal, signalSizeX * signalSizeZ * signalSizeY * sizeof(Complex),
        hipMemcpyDeviceToHost));

   // memcpy(h_result , buffer, C * W * sizeof(Complex));

    cout << "result\n";
    for (int z = 0; z < signalSizeZ; z++) {
        for (int y = 0; y < signalSizeY; y++) {
            for (int x = 0; x < signalSizeX; x++) {

                cout << "(" << h_result[x + signalSizeX * (y + signalSizeY * z)].x << "; " << h_result[x + signalSizeX * (y + signalSizeY * z)].y << ") ";
            }
            cout << "\n";

        }
        cout << "\n\n";
    }


Error:

    return cudaStatus;

}
// signal size is power of 2
// 2^34 ?
// two round 1d fft algo
hipError_t fft_1d(long long signalSize) {



    int X1 = (int)sqrt(signalSize);
    int X2 = signalSize / X1;
    // float2 2 * 4byte each element
    long long gpu_mem_size = 268435456; // bytes for 2GB
    long long gpu_mem_size_b = 268435456 * sizeof(Complex); // bytes for 2GB 

    // Allocate host and device memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size));

    // Initialize signal
    for (unsigned int i = 0; i < signalSize; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }


    //memory for twiddle
    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * X1 * X2));
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), gpu_mem_size));
    //compute twidlle factors
    // exp(2 * pi * (m - 1) * (j - 1) / n), m - ������, j - �������
    for (int i = 0; i < X1; i++) {
        for (int j = 0; j < X2; j++) {
            h_twiddle[i * X2 + j].x = real(polar(1.0, -2 * M_PI * i * j / signalSize));
            h_twiddle[i * X2 + j].y = imag(polar(1.0, -2 * M_PI * i * j / signalSize));
        }
    }
    



    C = X1;
    W = gpu_mem_size / C;

    // do i really need buffer ?
    Complex* buffer = reinterpret_cast<Complex*>(malloc(gpu_mem_size_b));


    int it_s = X2 / W;
    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_signal + (i * W) + (j * X2), W * sizeof(Complex));
            checkCudaErrors(hipMemcpyAsync(d_signal + (j * W), buffer + (j * W), W * sizeof(Complex), hipMemcpyHostToDevice));
        }

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //transfer twiddle
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_twiddle + (i * W) + (j * X2), W * sizeof(Complex));
        }
        checkCudaErrors(hipMemcpy(d_twiddle, buffer, gpu_mem_size_b,
            hipMemcpyHostToDevice));
        //make plan
        hipfftHandle plan_adv;

        int n[1] = { C };
        int inembed[] = { C };
        int onembed[] = { W };
        int istride = W;
        int idist = 1;
        int ostride = 1;
        int odist = C;
        int batch = W;

        //transposing layout
        checkCudaErrors(hipfftPlanMany(&plan_adv, 1, n, inembed, istride, idist,
                                        onembed, ostride, odist, HIPFFT_C2C,  batch));
        checkCudaErrors(hipfftExecC2C(plan_adv, reinterpret_cast<hipfftComplex*>(d_signal), 
                                     reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
        // twiddle factor multiplication
        dim3 threadsPerBlock(32, 32);
        dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (W + threadsPerBlock.y - 1) / threadsPerBlock.y);
        TwiddleMult<<<numBlocks, threadsPerBlock>>>(d_signal, d_twiddle);
        //transport to host

        checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b,
                hipMemcpyDeviceToHost));
        
        memcpy(h_result + (i * C * W), buffer, W * C * sizeof(Complex));
        
    }

    // fft size x2
    C = X2;
    W = gpu_mem_size / C;

    it_s = X1 / W;
    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_result + (i * W) + (j * X2), W * sizeof(Complex));
            checkCudaErrors(hipMemcpyAsync(d_signal + (j * W), buffer + (j * W), W * sizeof(Complex), hipMemcpyHostToDevice));
        }

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //make plan
        hipfftHandle plan_adv;
        int n[] = { W };
        int inembed[] = { C };
        int onembed[] = { C };
        int istride = C;
        int idist = 1;
        int ostride = C;
        int odist = 1;
        int batch = C;

        //checkCudaErrors(hipfftCreate(&plan_adv));
        //does it transpose? (stride, dist)
        checkCudaErrors(hipfftPlanMany(&plan_adv, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_adv, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));

        //transport to host

        checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b,
            hipMemcpyDeviceToHost));

        memcpy(h_result + (i * C * W), buffer, W * C);

    }

    
    return cudaStatus;
}

hipError_t fft_naive_3d(long long signalSizeY, long long signalSizeX, long long signalSizeZ){

    // Allocate host memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSizeY * signalSizeX * signalSizeZ));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSizeY * signalSizeX * signalSizeZ));

    // Initialize the memory for the signal
    for (unsigned int i = 0; i < signalSizeY * signalSizeX * signalSizeZ; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }

    // how much am i able to use (?)
    // float2 2 * 4byte each element
    long long gpu_mem_size = 268435456; // elements for 2 gb
    long long gpu_mem_size_b = 268435456 * sizeof(Complex); // bytes for 2GB 

    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size));

    if (signalSizeX * signalSizeY < gpu_mem_size) {
        fprintf(stderr, "Cannot use that algorithm, consider using Z - decomposition\n");
        goto Error;
    }
    //assume that X * Y can fit in gpu
    C = signalSizeZ;
    W = gpu_mem_size / C;
    Complex* buffer = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * C * W));


    int it_s = signalSizeY * signalSizeX / W;
    for (int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_signal + (i * W) + (j * signalSizeX * signalSizeY), W);
            checkCudaErrors(hipMemcpyAsync(d_signal + (j * W), buffer + (j * W), W, hipMemcpyHostToDevice));
        }

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //transfer twiddle


        //make plan for z fft
        hipfftHandle plan_advZ;

        int n[1] = { C };
        int inembed[] = { W };
        int onembed[] = { W };
        int istride = W;
        int idist = 1;
        int ostride = W;
        int odist = 1;
        int batch = W;

        //checkCudaErrors(hipfftCreate(&plan_adv));
        //it is transpose
        checkCudaErrors(hipfftPlanMany(&plan_advZ, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_advZ, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
        hipfftHandle plan_advX;

        // X fft
        n[0] = { (int)signalSizeX };
        inembed[0] = { W };
        onembed[0] = { W };
        istride = 1;
        idist = (int)signalSizeX;
        ostride = 1;
        odist = (int)signalSizeX;
        batch = C * W / signalSizeX;
        //transport to host


        checkCudaErrors(hipfftPlanMany(&plan_advX, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_advX, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
        checkCudaErrors(hipMemcpy(buffer, d_signal, gpu_mem_size_b,
            hipMemcpyDeviceToHost));
        memcpy(h_result + (i * C * W), buffer, C * W * sizeof(Complex));

    }
    // fft size x2
    C = signalSizeY;
    W = gpu_mem_size / C;



    it_s = signalSizeX * signalSizeZ / W;
    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        
         memcpy(buffer, h_result + (i * W * C), C * W * sizeof(Complex));
         checkCudaErrors(hipMemcpyAsync(d_signal, buffer, W * C, hipMemcpyHostToDevice));
        

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //make plan
        hipfftHandle plan_advY;
        int n[] = { C };
        int inembed[] = { W };
        int onembed[] = { W };
        int istride = signalSizeX;
        int idist = 1;
        int ostride = signalSizeX;
        int odist = 1;
        int batch = signalSizeX;

        //checkCudaErrors(hipfftCreate(&plan_adv));
        //does it transpose? (stride, dist)
        checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));

        for (int k = 0; k < W / signalSizeX; k++) {
            checkCudaErrors(hipfftExecC2C(plan_advY, d_signal + (k * signalSizeX * C),
                d_signal + (k * signalSizeX * C), HIPFFT_FORWARD));
        }

        //transport to host

        checkCudaErrors(hipMemcpy(buffer, d_signal, gpu_mem_size_b,
            hipMemcpyDeviceToHost));

        memcpy(h_result + (i * C * W), buffer, C * W * sizeof(Complex));

    }

    Error:

    return cudaStatus;
}
/*
hipError_t fft_3d(long long signalSizeX,long long signalSizeY, long long  signalSizeZ) {

    long long X = signalSizeX;
    long long Y = signalSizeY;
    long long Z = signalSizeZ;

    long long Z1 = (int)(sqrt(Z));
    long long Z2 = Z / Z1;

    long long  signalSize = X * Y * Z;

    // float2 2 * 4byte each element
    long long gpu_mem_size = 268435456; // elemnts for 2GB
    long long gpu_mem_size_b = 268435456 * sizeof(Complex); // bytes for 2GB 


    // Allocate host and device memory for the signal
    Complex* h_signal = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* h_result = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * signalSize));
    Complex* d_signal;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_signal), gpu_mem_size));


    srand(3);
    // Initialize signal
    for (unsigned int i = 0; i < signalSize; ++i) {
        h_signal[i].x = rand() / static_cast<float>(RAND_MAX);
        h_signal[i].y = 0;
    }


    //allocate memory for twiddle

    Complex* h_twiddle = reinterpret_cast<Complex*>(malloc(sizeof(Complex) * Z1 * Z2));
    Complex* d_twiddle;
    checkCudaErrors(hipMalloc(reinterpret_cast<void**>(&d_twiddle), gpu_mem_size));

    //compute twidlle factors
    // exp(2 * pi * (m - 1) * (j - 1) / n), m - ������, j - �������
    for (int i = 0; i < Z1; i++) {
        for (int j = 0; j < Z2; j++) {
            h_twiddle[i * Z2 + j].x = real(polar(1.0, -2 * M_PI * i * j / signalSize));
            h_twiddle[i * Z2 + j].y = imag(polar(1.0, -2 * M_PI * i * j / signalSize));
        }
    }
    


    C = Z1;
    W = gpu_mem_size / C;


    Complex* buffer = reinterpret_cast<Complex*>(gpu_mem_size_b);


    // X * Y fits into memory

    int it_s = X * Y * Z2 / W;
    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_signal + (i * W) + (j * X * Y * Z2), W);
            checkCudaErrors(hipMemcpyAsync(d_signal + (j * W), buffer + (j * W), W, hipMemcpyHostToDevice));
        }

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //transfer twiddle
        
        memcpy(buffer, h_twiddle + (i * W/(X * Y)) , C * W / (X * Y)); 
        
        checkCudaErrors(hipMemcpy(d_twiddle, buffer, gpu_mem_size_b,
            hipMemcpyHostToDevice));
        //make plan for d(Z1, XYZ2, XY)
        hipfftHandle plan_advZ1;

        int n[1] = { C };
        int inembed[] = { C };
        int onembed[] = { W };
        int istride = W;
        int idist = 1;
        int ostride = X * Y;
        int odist = 1;
        int batch = W;

        // transpose by advanced layout
        checkCudaErrors(hipfftPlanMany(&plan_advZ1, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_advZ1, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));
        // have to mult by twiddle factor
        int h_XY = X * Y;
        checkCudaErrors(hipMemset(&XY, 0, sizeof(int)));
        checkCudaErrors(hipMemcpyFromSymbol(&h_XY, HIP_SYMBOL(&XY), sizeof(int)));

        dim3 threadsPerBlock(32, 32);
        dim3 numBlocks((C + threadsPerBlock.x - 1) / threadsPerBlock.x,
            (W + threadsPerBlock.y - 1) / threadsPerBlock.y);
        TwiddleMult << <numBlocks, threadsPerBlock >> > (d_signal, d_twiddle, XY);
        //transport to host

        checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b,
            hipMemcpyDeviceToHost));

        memcpy(h_result + (i * C * W), buffer, W * C);
        //make plan for d(Z1, XYZ2, XY)

        hipfftHandle plan_advY;

        n[0] = Y ;
        inembed[0] = C;
        onembed[0] = W;
        istride = X;
        idist = 1;
        ostride = X;
        odist = 1;
        batch = W * C / Y;

        // transpose by advanced layout
        checkCudaErrors(hipfftPlanMany(&plan_advY, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_advY, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));

    }
    // fft size x2
    C = X2;
    W = gpu_mem_size / C;



    it_s = X1 / W;
    for (unsigned int i = 0; i < it_s; i += 1) {
        //transfer data to gpu
        for (unsigned int j = 0; j < C; j++) {
            memcpy(buffer + (j * W), h_result + (i * W) + (j * X2), W);
            checkCudaErrors(hipMemcpyAsync(d_signal + (j * W), buffer + (j * W), W, hipMemcpyHostToDevice));
        }

        //checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b, 
        //                           hipMemcpyHostToDevice));

        //make plan
        hipfftHandle plan_adv;
        int n[] = { W };
        int inembed[] = { C };
        int onembed[] = { C };
        int istride = C;
        int idist = 1;
        int ostride = C;
        int odist = 1;
        int batch = C;

        //checkCudaErrors(hipfftCreate(&plan_adv));
        //does it transpose? (stride, dist)
        checkCudaErrors(hipfftPlanMany(&plan_adv, 1, n, inembed, istride, idist,
            onembed, ostride, odist, HIPFFT_C2C, batch));
        checkCudaErrors(hipfftExecC2C(plan_adv, reinterpret_cast<hipfftComplex*>(d_signal),
            reinterpret_cast<hipfftComplex*>(d_signal), HIPFFT_FORWARD));

        //transport to host

        checkCudaErrors(hipMemcpy(d_signal, buffer, gpu_mem_size_b,
            hipMemcpyDeviceToHost));

        memcpy(h_result + (i * C * W), buffer, W * C);

    }


    return cudaStatus;
}
*/
////////////////////////////////////////////////////////////////////////////////
// Complex operations
////////////////////////////////////////////////////////////////////////////////

// Complex addition
static __device__ __host__ inline Complex ComplexAdd(Complex a, Complex b) {
    Complex c;
    c.x = a.x + b.x;
    c.y = a.y + b.y;
    return c;
}

// Complex scale
static __device__ __host__ inline Complex ComplexScale(Complex a, float s) {
    Complex c;
    c.x = s * a.x;
    c.y = s * a.y;
    return c;
}

// Complex multiplication
static __device__ __host__ inline Complex ComplexMul(Complex a, Complex b) {
    Complex c;
    c.x = a.x * b.x - a.y * b.y;
    c.y = a.x * b.y + a.y * b.x;
    return c;
}

// Complex pointwise multiplication
static __global__ void TwiddleMult(Complex* X, Complex* twiddle) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < C && j < W)
        X[j*C + i] = ComplexMul(X[j * C + i], twiddle[i*W + j]);
}
static __global__ void TwiddleMult(Complex* X, Complex* twiddle, int XY) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < C && j < W)
        X[j * C + i] = ComplexMul(X[j * C + i], twiddle[i * W/XY + j]);
}